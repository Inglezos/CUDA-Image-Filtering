
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void Babis_Kernel(double const *A, double  *B, double const *G, int n, int m, int patchSize_x, int patchSize_y, double filtSigma)
{
  int x,y,area_x,area_y;
  double norm,w_temp,diff=0,W=0,Products=0;
 

  // Set pixel coordinates

  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  
  // For every pixel we check every other pixel neighbor, if we are inside the table limits

  if( ( i>=patchSize_x ) && ( j>=patchSize_y ) && ( i<=(n-patchSize_x) -1 ) && ( j<=(m-patchSize_x) -1 ) )
  { 
    for(x=patchSize_x; x< (n-patchSize_x) ; x++)
    {
	   for(y=patchSize_y; y< (m-patchSize_y) ; y++)
        {
		   norm=0; 
		   w_temp=0;
		   
		// i,j indicate the coordinates of the current thread, while x,y indicate the coordinates of every other pixel-neighbor in the table/image and x_area,y_area indicate 
		// the area around the neighbor, for example 3x3, 5x5, 7x7.

		   for(area_x=-patchSize_x; area_x<=patchSize_x; area_x++)
             {
		      for(area_y=-patchSize_y; area_y<=patchSize_y; area_y++)
                {

				  diff=abs(A[(i+area_x)*m+(j+area_y)]-A[(x+area_x)*m+(y+area_y)]);

				  diff=diff*G[(area_x+patchSize_x)*((patchSize_y*2)+1)+(area_y+patchSize_y)];

				  diff=diff*diff;

				  norm+=diff;  
			 }
		    }

		    w_temp=exp(-norm/filtSigma);
		    W+=w_temp;
		    Products+=w_temp*A[x*m+y];
	    }
	}
  
     B[(i-patchSize_x)*(m-(2*patchSize_y)) + (j-patchSize_y)] = Products/W;
  }
}

